#include <stdio.h>
#include <hip/hip_runtime.h>

// __ is pronounced "dunder" (double underscore)
__global__ void square(float *d_out, float *d_in)
{
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f;
}

int main(int argc, char **argv)
{
    const int ARRAY_SIZE = 64;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // Declare and initialize CPU arrays
    float h_in[ARRAY_SIZE];
    float h_out[ARRAY_SIZE];
    for (int i = 0; i < ARRAY_SIZE; i++)
    {
        h_in[i] = float(i);
    }

    // Declare and allocate GPU memory.
    float *d_in;
    float *d_out;

    hipMalloc(
        (void **)&d_in,
        ARRAY_BYTES);   // numbers of bytes to allocate
    hipMalloc((void **)&d_out, ARRAY_BYTES);

    // Copy array to GPU.
    hipMemcpy(d_in,                    // reference on the cpu side to memory allocated on the gpu
               h_in,                    // reference on the cpu side
               ARRAY_BYTES,             // how many bytes to copy from h_in to d_in
               hipMemcpyHostToDevice); // direction

    // Launch the kernel.
    square<<<1, ARRAY_SIZE>>>(d_out, d_in);

    // Copy results back from GPU.
    hipMemcpy(h_out,                   // destination
               d_out,                   // source
               ARRAY_BYTES,             
               hipMemcpyDeviceToHost);

    // Release GPU memory.
    hipFree(d_in);
    hipFree(d_out);

    // Print the results.
    for (int i = 0; i < ARRAY_SIZE; i++)
    {
        printf("%f", h_out[i]);
    }
    printf("\n");

    return 0;
}
